
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdio.h>

#define USE_STREAMS

// modifiable
typedef float ft;
const int chunks = 64;
const size_t ds = 1024*1024*chunks;
const int count = 22;
const int num_streams = 8;

// not modifiable
const float sqrt_2PIf = 2.5066282747946493232942230134974f;
const double sqrt_2PI = 2.5066282747946493232942230134974;
__device__ float gpdf(float val, float sigma) {
  return expf(-0.5f * val * val) / (sigma * sqrt_2PIf);
}

__device__ double gpdf(double val, double sigma) {
  return exp(-0.5 * val * val) / (sigma * sqrt_2PI);
}

// compute average gaussian pdf value over a window around each point
__global__ void gaussian_pdf(const ft * __restrict__ x, ft * __restrict__ y, const ft mean, const ft sigma, const int n) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    ft in = x[idx] - (count / 2) * 0.01f;
    ft out = 0;
    for (int i = 0; i < count; i++) {
      ft temp = (in - mean) / sigma;
      out += gpdf(temp, sigma);
      in += 0.01f;
    }
    y[idx] = out / count;
  }
}

// error check macro
#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)

// host-based timing
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start) {
  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

int main() {
  ft *h_x, *d_x, *h_y, *h_y1, *d_y;
  hipHostAlloc(&h_x,  ds*sizeof(ft), hipHostMallocDefault);
  hipHostAlloc(&h_y,  ds*sizeof(ft), hipHostMallocDefault);
  hipHostAlloc(&h_y1, ds*sizeof(ft), hipHostMallocDefault);
  hipMalloc(&d_x, ds*sizeof(ft));
  hipMalloc(&d_y, ds*sizeof(ft));
  cudaCheckErrors("allocation error");

  hipStream_t streams[num_streams];
  for (int i = 0; i < num_streams; i++) {
    hipStreamCreate(&streams[i]);
  }
  cudaCheckErrors("stream creation error");

  gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x, d_y, 0.0, 1.0, ds); // warm-up

  for (size_t i = 0; i < ds; i++) {
    h_x[i] = rand() / (ft)RAND_MAX;
  }
  hipDeviceSynchronize();

  hipEvent_t start, stop;

  float float_time = 0;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  hipMemcpy(d_x, h_x, ds * sizeof(ft), hipMemcpyHostToDevice);
  gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x, d_y, 0.0, 1.0, ds);
  hipMemcpy(h_y1, d_y, ds * sizeof(ft), hipMemcpyDeviceToHost);
  cudaCheckErrors("non-streams execution error");

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&float_time, start, stop);

  printf("non-streams elapsed time: %f\n",  float_time / (float)100);

#ifdef USE_STREAMS
  hipMemset(d_y, 0, ds * sizeof(ft));

  float float_time_streams = 0;
  hipEventRecord(start);

  for (int i = 0; i < chunks; i++) { //depth-first launch
    hipMemcpyAsync(d_x + i * (ds / chunks), h_x + i * (ds / chunks), (ds / chunks) * sizeof(ft), hipMemcpyHostToDevice, streams[i % num_streams]);
    gaussian_pdf<<<((ds / chunks) + 255) / 256, 256, 0, streams[i % num_streams]>>>(d_x + i * (ds / chunks), d_y + i * (ds / chunks), 0.0, 1.0, ds / chunks);
    hipMemcpyAsync(h_y + i * (ds / chunks), d_y + i * (ds / chunks), (ds / chunks) * sizeof(ft), hipMemcpyDeviceToHost, streams[i % num_streams]);
  }
  hipDeviceSynchronize();
  cudaCheckErrors("streams execution error");

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&float_time_streams, start, stop);

  for (int i = 0; i < ds; i++) {
    if (h_y[i] != h_y1[i]) {
      std::cout << "mismatch at " << i << " was: " << h_y[i] << " should be: " << h_y1[i] << std::endl;
      return -1;
    }
  }

  printf("streams elapsed time: %f\n", float_time_streams / (float)100);
#endif

  return 0;
}
